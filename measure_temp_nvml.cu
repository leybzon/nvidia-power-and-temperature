#include <hip/hip_runtime.h>
#include <nvml.h>
#include <iostream>
#include <chrono>
#include <thread>

// Kernel function to perform matrix multiplication
__global__ void matMulKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

void checkNvmlErrors(nvmlReturn_t result) {
    if (result != NVML_SUCCESS) {
        std::cerr << "NVML Error: " << nvmlErrorString(result) << std::endl;
        exit(-1);
    }
}

int main() {
    int N = 2048; // Increased matrix size to better utilize the GPU
    size_t bytes = N * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float *)malloc(bytes);
    float *h_B = (float *)malloc(bytes);
    float *h_C = (float *)malloc(bytes);

    // Initialize host matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc(&d_A, bytes));
    checkCudaErrors(hipMalloc(&d_B, bytes));
    checkCudaErrors(hipMalloc(&d_C, bytes));

    // Copy data to device
    checkCudaErrors(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    // Define grid and block dimensions
    dim3 block(32, 32); // Larger block size to maximize utilization
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    // Initialize NVML
    checkNvmlErrors(nvmlInit());

    // Get the handle for the first GPU
    nvmlDevice_t device;
    checkNvmlErrors(nvmlDeviceGetHandleByIndex(0, &device));

    // Measure the temperature, power consumption, and calculation rate over 10 minutes
    auto start = std::chrono::high_resolution_clock::now();
    auto end = start;
    auto next_temp_check = start + std::chrono::seconds(10);
    int matrixCalculations = 0;

    do {
        // Launch the kernel
        matMulKernel<<<grid, block>>>(d_A, d_B, d_C, N);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        matrixCalculations++;

        end = std::chrono::high_resolution_clock::now();
        if (end >= next_temp_check) {
            unsigned int temp;
            unsigned int power;

            checkNvmlErrors(nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temp));
            checkNvmlErrors(nvmlDeviceGetPowerUsage(device, &power)); // Power in milliwatts

            double elapsedTime = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
            double calculationsPerSecond = matrixCalculations / elapsedTime;

            std::cout << "Time: " << elapsedTime << " seconds, "
                      << "Current GPU temperature: " << temp << " C, "
                      << "Power consumption: " << power / 1000.0 << " W, "
                      << "Matrix calculations per second: " << calculationsPerSecond << std::endl;

            next_temp_check = end + std::chrono::seconds(10);
        }
    } while (std::chrono::duration_cast<std::chrono::minutes>(end - start).count() < 10);

    // Get final temperature and power consumption
    unsigned int tempAfter;
    unsigned int powerAfter;

    checkNvmlErrors(nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &tempAfter));
    checkNvmlErrors(nvmlDeviceGetPowerUsage(device, &powerAfter));

    std::cout << "Final GPU temperature: " << tempAfter << " C" << std::endl;
    std::cout << "Final GPU power consumption: " << powerAfter / 1000.0 << " W" << std::endl;

    // Cleanup NVML
    checkNvmlErrors(nvmlShutdown());

    // Copy result back to host (optional)
    checkCudaErrors(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << "Completed 10 minutes of GPU load for temperature and power measurement." << std::endl;
    return 0;
}
